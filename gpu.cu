#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "parameters.hh"
#include "gpu.hh"

#define DEBUG

#define test 0

#define ABORT_ON_ERROR(f)												\
	do {																\
		hipError_t status_ = (f);										\
		if (status_ != hipSuccess) {									\
			fprintf(stderr, "Fatal error: " #f ": %s\n(%s:%d)\nABORTING\n", \
					hipGetErrorString(status_),						\
					__FILE__, __LINE__);								\
			hipDeviceReset() ;											\
			exit(0);													\
		}																\
	} while (0)

#define WARNING_ON_ERROR(f)												\
	do {																\
		hipError_t status_ = (f);										\
		if (status_ != hipSuccess) {									\
			fprintf(stderr, "Error: " #f ": %s\n(%s:%d)\n",				\
					hipGetErrorString(status_),						\
					__FILE__, __LINE__);								\
		}																\
	} while (0)

hipStream_t streams[GPU_STREAMS];

__align__(32) __constant__ __device__
uint32_t packets[GPU_STREAMS][PACKETS_BATCH_SIZE*GPU_FILTER_WORDS];

__device__ bool a_complement_not_subset_of_b(uint32_t a, uint32_t b) { 
	return ((a | b) != (~0U));
}

#if 0
#define TI_TREE_OFFSET 13
#define TI_INTERFACE_MASK (((uint16_t)1 << TI_TREE_OFFSET) - 1)
#else
static const unsigned int TI_TREE_OFFSET = 13;
static const uint16_t TI_INTERFACE_MASK = (1 << TI_TREE_OFFSET) - 1;
#endif

__device__ uint16_t ti_tree(uint16_t ti_pair) {
	return ti_pair >> TI_TREE_OFFSET;
}

__device__ uint16_t ti_interface(uint16_t ti_pair) {
	return ti_pair & TI_INTERFACE_MASK;
}

#if WITH_GPU_FAST_KERNEL

__global__ void fast_kernel(uint32_t * fib, unsigned int fib_size, 
							uint16_t * ti_table, unsigned int * ti_indexes,  
							uint16_t * query_ti_table ,  unsigned int batch_size, 
							ifx_result_t * results,  
							unsigned int stream_id)
{
	unsigned int t1 = (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y) ;
	unsigned int id = t1 + threadIdx.x;

	if(id >= fib_size)
		return;

	unsigned int f_id = GPU_FILTER_WORDS*(t1) + threadIdx.x;
		
	unsigned int f[GPU_FILTER_WORDS];
	f[0]=fib[f_id];
	f[1]=fib[f_id+32];
	f[2]=fib[f_id+64];
	f[3]=fib[f_id+96];
	f[4]=fib[f_id+128];
	f[5]=fib[f_id+160];

	for(unsigned int j = 0; batch_size > 0; --batch_size, j += GPU_FILTER_WORDS, ++query_ti_table) {

		if (a_complement_not_subset_of_b(f[5], packets[stream_id][j+6]))
		    continue;
		if (a_complement_not_subset_of_b(f[4], packets[stream_id][j+4]))
		    continue;
		if (a_complement_not_subset_of_b(f[3], packets[stream_id][j+3]))
		    continue;
		if (a_complement_not_subset_of_b(f[2], packets[stream_id][j+2]))
		    continue;
		if (a_complement_not_subset_of_b(f[1], packets[stream_id][j+1]))
		    continue;
		if (a_complement_not_subset_of_b(f[0], packets[stream_id][j]))
		    continue;

//		printf("matched!\n");

		unsigned int ti_index = ti_indexes[id];
		for(unsigned int i = ti_table[ti_index]; i > 0; --i) {
			// TODO: document these cryptic operations.
			uint16_t xor_tmp = *query_ti_table ^ ti_table[ti_index + i];
			if ((xor_tmp <= tree_interface_pair::IFX_MASK) && (xor_tmp != 0))
				results[(i * INTERFACES) + ((ti_table[ti_index + i]) & 0x1FFF)] = 1;
		}
	}
}

#else // no WITH_GPU_FAST_KERNEL

__global__ void minimal_kernel(uint32_t * fib, unsigned int fib_size, 
							   uint16_t * ti_table, unsigned int * ti_indexes,  
							   uint16_t * query_ti_table ,  unsigned int batch_size, 
							   ifx_result_t * results,
							   unsigned int stream_id) {

	unsigned int id = (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y) + threadIdx.x;

	if(id >= fib_size)
		return;

	fib += (id * GPU_FILTER_WORDS);
	uint32_t f[GPU_FILTER_WORDS];

	for(unsigned int i = 0; i < GPU_FILTER_WORDS; ++i) 
		f[i] = fib[i];

	uint32_t * p = packets[stream_id] + 5;
	for(unsigned int pi = 0; pi < batch_size; ++pi) {
		if (a_complement_not_subset_of_b(f[5], *p)) {
			p += (0 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[4], *p)) {
			p += (1 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[3], *p)) {
			p += (2 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[2], *p)) {
			p += (3 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[1], *p)) {
			p += (4 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[0], *p)) {
			p += (5 + GPU_FILTER_WORDS);
			continue;
		}
		p += (5 + GPU_FILTER_WORDS);

		unsigned int ti_index = ti_indexes[id];
		for(unsigned int i = ti_table[ti_index]; i > 0; --i) {
			// TODO: document these cryptic operations.
			uint16_t ti_xor = query_ti_table[pi] ^ ti_table[ti_index + i];
			if ((ti_xor <= 0x1FFF) && (ti_xor != 0)) {
				results[pi*INTERFACES + ((ti_table[ti_index + i]) & 0x1FFF)] = 1;
			}
		}
	}
}
#endif

void gpu::initialize() {
	ABORT_ON_ERROR(hipSetDevice(0));
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipDeviceSynchronize());
	for(unsigned int i = 0; i < GPU_STREAMS; ++i)
		ABORT_ON_ERROR(hipStreamCreate(streams + i));
}

void gpu::mem_info(gpu_mem_info * mi) {
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipMemGetInfo(&(mi->free), &(mi->total)));
}

void gpu::async_copy_packets(unsigned int * host_packets, unsigned int size , unsigned int stream) {
	ABORT_ON_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(packets), host_packets, size*GPU_FILTER_WORDS*sizeof(unsigned int), stream*PACKETS_BATCH_SIZE*GPU_FILTER_WORDS*sizeof(unsigned int), hipMemcpyHostToDevice, streams[stream]));
}

// allocates memory for a table on the device of the given byte size
//
void * gpu::allocate_generic(unsigned int size) {
	void * dev_table = 0;
	ABORT_ON_ERROR(hipMalloc(&dev_table, size));
	return dev_table; 
}

// allocates memory for a table on the device of the given byte size
// and then copies the content from the host table into the device
// table
//
void * gpu::allocate_and_copy_generic(void * host_table, unsigned int size) {
	void * dev_table = 0;
	ABORT_ON_ERROR(hipMalloc(&dev_table, size)); 
	ABORT_ON_ERROR(hipMemcpy(dev_table, host_table, size, hipMemcpyHostToDevice));
	return dev_table; 
}

void gpu::async_copy(void * host_src, void * dev_dst, unsigned int size, unsigned int stream_id) {
	ABORT_ON_ERROR(hipMemcpyAsync(dev_dst, host_src, size, hipMemcpyHostToDevice, streams[stream_id]));
}

 // this is useful for clearing the dev_res (interfaces) to 0 before
 // calling the kernel
void gpu::async_set_zero(void * dev_array, unsigned int size, unsigned int stream_id) {
	ABORT_ON_ERROR(hipMemsetAsync(dev_array, 0, size, streams[stream_id]));
}


void gpu::async_get_results(ifx_result_t * host_results, ifx_result_t * dev_results, 
							unsigned int size, unsigned int stream) {
	ABORT_ON_ERROR(hipMemcpyAsync(host_results, dev_results, size * INTERFACES * sizeof(ifx_result_t), hipMemcpyDeviceToHost, streams[stream]));
}

void gpu::get_results(ifx_result_t * host_results, ifx_result_t * dev_results, unsigned int size) {
	ABORT_ON_ERROR(hipMemcpy(host_results, dev_results, size * INTERFACES * sizeof(ifx_result_t), hipMemcpyDeviceToHost));
}

void gpu::synchronize_device() {
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipDeviceSynchronize());
}

void gpu::synchronize_stream(unsigned int stream) {
	ABORT_ON_ERROR(hipStreamSynchronize(streams[stream]));
}

#if WITH_PINNED_HOST_MEMORY
void * gpu::allocate_host_pinned_generic(unsigned int size) {
	void * host_array_pinned;
	ABORT_ON_ERROR(hipHostMalloc(&host_array_pinned, size));
	return host_array_pinned;
}
#endif

static const dim3 BLOCK_DIMS(GPU_BLOCK_DIM_X, GPU_BLOCK_DIM_Y);

void gpu::run_kernel(uint32_t * fib, unsigned int fib_size, 
					 uint16_t * ti_table, unsigned int * ti_indexes, 
					 uint16_t * query_ti_table, unsigned int batch_size, 
					 ifx_result_t * results, 
					 unsigned int stream) {

	unsigned int gridsize = fib_size/GPU_BLOCK_SIZE;
	if ((fib_size % GPU_BLOCK_SIZE) != 0)
		++gridsize;

#if WITH_GPU_FAST_KERNEL
	fast_kernel<<< gridsize, BLOCK_DIMS, 0, streams[stream] >>> (fib, 
																 fib_size,
																 ti_table, 
																 ti_indexes, 
																 query_ti_table, 
																 batch_size, 
																 results, 
																 stream);
#else
	minimal_kernel<<<gridsize, BLOCK_DIMS, 0, streams[stream] >>> (fib, 
																   fib_size,
																   ti_table, 
																   ti_indexes, 
																   query_ti_table,
																   batch_size,
																   results,
																   stream);
#endif
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Fatal error: run_kernel: %s\n(%s:%d)\nABORTING\n", 
				hipGetErrorString(status), __FILE__, __LINE__);
		hipDeviceReset() ; 
		exit(0); 
	} 
}

void gpu::shutdown() {
	// TODO: deallocate 
	for(unsigned int i = 0; i < GPU_STREAMS; ++i)
		ABORT_ON_ERROR(hipStreamDestroy(streams[i]));
	hipDeviceReset();
}

void gpu::release_memory(void * p) {
	ABORT_ON_ERROR(hipFree(p)); 
}

void gpu::release_pinned_memory(void * p) {
	ABORT_ON_ERROR(hipHostFree(p)); 
}

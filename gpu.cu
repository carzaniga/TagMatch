#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "parameters.hh"
#include "gpu.hh"

#define DEBUG

#define test 0

#define ABORT_ON_ERROR(f)												\
	do {																\
		hipError_t status_ = (f);										\
		if (status_ != hipSuccess) {									\
			fprintf(stderr, "Fatal error: " #f ": %s\n(%s:%d)\nABORTING\n", \
					hipGetErrorString(status_),						\
					__FILE__, __LINE__);								\
			hipDeviceReset() ;											\
			exit(0);													\
		}																\
	} while (0)

#define WARNING_ON_ERROR(f)												\
	do {																\
		hipError_t status_ = (f);										\
		if (status_ != hipSuccess) {									\
			fprintf(stderr, "Error: " #f ": %s\n(%s:%d)\n",				\
					hipGetErrorString(status_),						\
					__FILE__, __LINE__);								\
		}																\
	} while (0)

hipStream_t streams[GPU_STREAMS];

__align__(32) __constant__ __device__
uint32_t packets[GPU_STREAMS][PACKETS_BATCH_SIZE*GPU_FILTER_WORDS];

__device__ bool a_complement_not_subset_of_b(uint32_t a, uint32_t b) { 
	return ((a | b) != (~0U));
}

__global__ void minimal_kernel(uint32_t * fib, unsigned int fib_size, 
							   uint16_t * ti_table, unsigned int * ti_indexes,  
							   uint16_t * query_ti_table ,  unsigned int batch_size, 
							   ifx_result_t * results,
							   unsigned int stream_id) {

	unsigned int id = (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y) + threadIdx.x;

	if(id >= fib_size)
		return;

	fib += (id * GPU_FILTER_WORDS);

	uint32_t f[GPU_FILTER_WORDS];

	for(unsigned int i = 0; i < GPU_FILTER_WORDS; ++i) 
		f[i] = fib[i];

#if 0
	uint32_t * p = packets[stream_id] + 5;
	for(unsigned int pi = 0; pi < batch_size; ++pi) {
		if (a_complement_not_subset_of_b(f[5], *p)) {
			p += (0 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[4], *p)) {
			p += (1 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[3], *p)) {
			p += (2 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[2], *p)) {
			p += (3 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[1], *p)) {
			p += (4 + GPU_FILTER_WORDS);
			continue;
		}
		--p;
		if (a_complement_not_subset_of_b(f[0], *p)) {
			p += (5 + GPU_FILTER_WORDS);
			continue;
		}
		p += (5 + GPU_FILTER_WORDS);

		unsigned int ti_index = ti_indexes[id];
		for(unsigned int i = ti_table[ti_index]; i > 0; --i) {
			// TODO: document these cryptic operations.
			uint16_t ti_xor = query_ti_table[pi] ^ ti_table[ti_index + i];
			if ((ti_xor <= 0x1FFF) && (ti_xor != 0)) {
				results[pi*INTERFACES + ((ti_table[ti_index + i]) & 0x1FFF)] = 1;
			}
		}
	}
#else
	for(unsigned int pi = 0; pi < batch_size; ++pi) {
		if (a_complement_not_subset_of_b(f[0], packets[stream_id][pi*GPU_FILTER_WORDS + 0]))
			continue;
		if (a_complement_not_subset_of_b(f[1], packets[stream_id][pi*GPU_FILTER_WORDS + 1]))
			continue;
		if (a_complement_not_subset_of_b(f[2], packets[stream_id][pi*GPU_FILTER_WORDS + 2]))
			continue;
		if (a_complement_not_subset_of_b(f[3], packets[stream_id][pi*GPU_FILTER_WORDS + 3]))
			continue;
		if (a_complement_not_subset_of_b(f[4], packets[stream_id][pi*GPU_FILTER_WORDS + 4]))
			continue;
		if (a_complement_not_subset_of_b(f[5], packets[stream_id][pi*GPU_FILTER_WORDS + 5]))
			continue;

		unsigned int ti_index = ti_indexes[id];
		for(unsigned int i = ti_table[ti_index]; i > 0; --i) {
			// TODO: document these cryptic operations.
			uint16_t ti_xor = query_ti_table[pi] ^ ti_table[ti_index + i];
			if ((ti_xor < (0x0001 << 13)) && (ti_xor != 0)) {
				results[pi*INTERFACES + ((ti_table[ti_index + i]) & (0xFFFF >> 3))] = 1;
			}
		}
	}
#endif
}

void gpu::initialize() {
	ABORT_ON_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	ABORT_ON_ERROR(hipSetDevice(0));
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipDeviceSynchronize());
	for(unsigned int i = 0; i < GPU_STREAMS; ++i)
		ABORT_ON_ERROR(hipStreamCreate(streams + i));
}

void gpu::mem_info(gpu_mem_info * mi) {
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipMemGetInfo(&(mi->free), &(mi->total)));
}

void gpu::async_copy_packets(unsigned int * host_packets, unsigned int size , unsigned int stream) {
	ABORT_ON_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(packets), host_packets, size*GPU_FILTER_WORDS*sizeof(unsigned int), stream*PACKETS_BATCH_SIZE*GPU_FILTER_WORDS*sizeof(unsigned int), hipMemcpyHostToDevice, streams[stream]));
}

// allocates memory for a table on the device of the given byte size
//
void * gpu::allocate_generic(unsigned int size) {
	void * dev_table = 0;
	ABORT_ON_ERROR(hipMalloc(&dev_table, size));
	return dev_table; 
}

// allocates memory for a table on the device of the given byte size
// and then copies the content from the host table into the device
// table
//
void * gpu::allocate_and_copy_generic(void * host_table, unsigned int size) {
	void * dev_table = 0;
	ABORT_ON_ERROR(hipMalloc(&dev_table, size)); 
	ABORT_ON_ERROR(hipMemcpy(dev_table, host_table, size, hipMemcpyHostToDevice));
	return dev_table; 
}

void gpu::async_copy(void * host_src, void * dev_dst, unsigned int size, unsigned int stream_id) {
	ABORT_ON_ERROR(hipMemcpyAsync(dev_dst, host_src, size, hipMemcpyHostToDevice, streams[stream_id]));
}

 // this is useful for clearing the dev_res (interfaces) to 0 before
 // calling the kernel
void gpu::async_set_zero(void * dev_array, unsigned int size, unsigned int stream_id) {
	ABORT_ON_ERROR(hipMemsetAsync(dev_array, 0, size, streams[stream_id]));
}


void gpu::async_get_results(ifx_result_t * host_results, ifx_result_t * dev_results, 
							unsigned int size, unsigned int stream) {
	ABORT_ON_ERROR(hipMemcpyAsync(host_results, dev_results, size * INTERFACES * sizeof(ifx_result_t), hipMemcpyDeviceToHost, streams[stream]));
}

void gpu::get_results(ifx_result_t * host_results, ifx_result_t * dev_results, unsigned int size) {
	ABORT_ON_ERROR(hipMemcpy(host_results, dev_results, size * INTERFACES * sizeof(ifx_result_t), hipMemcpyDeviceToHost));
}

void gpu::synchronize_device() {
	ABORT_ON_ERROR(hipDeviceSynchronize());
	ABORT_ON_ERROR(hipDeviceSynchronize());
}

void gpu::synchronize_stream(unsigned int stream) {
	ABORT_ON_ERROR(hipStreamSynchronize(streams[stream]));
}

void * gpu::allocate_host_pinned_generic(unsigned int size) {
	void * host_array_pinned;
	ABORT_ON_ERROR(hipHostMalloc(&host_array_pinned, size));
	return host_array_pinned;
}

static const dim3 BLOCK_DIMS(GPU_BLOCK_DIM_X, GPU_BLOCK_DIM_Y);

void gpu::run_kernel(uint32_t * fib, unsigned int fib_size, 
					 uint16_t * ti_table, unsigned int * ti_indexes, 
					 uint16_t * query_ti_table, unsigned int batch_size, 
					 ifx_result_t * results, 
					 unsigned int stream) {

	unsigned int gridsize = fib_size/GPU_BLOCK_SIZE;
	if ((fib_size % GPU_BLOCK_SIZE) != 0)
		++gridsize;

	minimal_kernel<<<gridsize, BLOCK_DIMS, 0, streams[stream] >>> (fib, 
																   fib_size,
																   ti_table, 
																   ti_indexes, 
																   query_ti_table,
																   batch_size,
																   results,
																   stream);

	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Fatal error: run_kernel: %s\n(%s:%d)\nABORTING\n", 
				hipGetErrorString(status), __FILE__, __LINE__);
		hipDeviceReset() ; 
		exit(0); 
	} 
}

void gpu::shutdown() {
	// TODO: deallocate 
	for(unsigned int i = 0; i < GPU_STREAMS; ++i)
		ABORT_ON_ERROR(hipStreamDestroy(streams[i]));
	hipDeviceReset();
}

void gpu::release_memory(void * p) {
	ABORT_ON_ERROR(hipFree(p)); 
}

void gpu::release_pinned_memory(void * p) {
	ABORT_ON_ERROR(hipHostFree(p)); 
}
